#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "SyncedMemory.h"
#include "lab1.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

int main(int argc, char **argv)
{
	Lab1VideoGenerator g;
	Lab1VideoInfo i;

	g.get_info(i);
	if (i.w == 0 or i.h == 0 or i.n_frame == 0 or i.fps_n == 0 or i.fps_d == 0) {
		puts("Cannot be zero");
		abort();
	} else if (i.w%2 != 0 or i.h%2 != 0) {
		puts("Only even frame size is supported");
		abort();
	}
	unsigned FRAME_SIZE = i.w*i.h*3/2;
	MemoryBuffer<uint8_t> frameb(FRAME_SIZE);
	auto frames = frameb.CreateSync(FRAME_SIZE);
	FILE *fp = fopen("result.y4m", "wb");
	fprintf(fp, "YUV4MPEG2 W%d H%d F%d:%d Ip A1:1 C420\n", i.w, i.h, i.fps_n, i.fps_d);

	for (unsigned j = 0; j < i.n_frame; ++j) {
		fputs("FRAME\n", fp);
		g.Generate(frames.get_gpu_wo());
		fwrite(frames.get_cpu_ro(), sizeof(uint8_t), FRAME_SIZE, fp);
	}

	fclose(fp);
	return 0;
}
