#include "hip/hip_runtime.h"
#include "lab1.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <algorithm>
#define maxStack 512
static const unsigned W = 960;
static const unsigned H = 960;
static const unsigned NFRAME = W;

typedef struct {
	int32_t l, r, pivotPos, rPos, pivot;
} Stack;

Stack stack[maxStack];
int stackF = -1;
uint32_t V[W];

uint32_t *bufferV;
Stack *bufferStack;

struct Lab1VideoGenerator::Impl {
	int t = 0;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

void myStackPush(uint32_t l, uint32_t r, uint32_t pivotPos, uint32_t rPos, uint32_t pivot){	
	stackF++;
	stack[stackF].l = l;
	stack[stackF].r = r;
	stack[stackF].pivot = pivot;
	stack[stackF].pivotPos = pivotPos;
	stack[stackF].rPos = rPos;
}

void myPreprocessing(){
	srand(time(NULL));
	for (int i=0; i<W; i++)
		V[i] = i;
	std::random_shuffle(V, V+W);
	myStackPush(0, W-1, 0, W-1, V[0]);

	hipMalloc(&bufferV, W*sizeof(uint32_t));
	hipMalloc(&bufferStack, maxStack*sizeof(Stack));
}

void myQuickSort(){
	if (stackF < 0) return;
	int32_t l, r, pivot, pivotPos, rPos;
	l = stack[stackF].l;
	r = stack[stackF].r;
	pivot = stack[stackF].pivot;
	pivotPos = stack[stackF].pivotPos;
	rPos = stack[stackF].rPos;
	stackF--;

	if(l >= r) return;

	if (pivotPos >= rPos){
		myStackPush(l, pivotPos-1, l, pivotPos-1, V[l]);
		myStackPush(pivotPos+1, r, pivotPos+1, r, V[pivotPos+1]);
		return;
	}
	if (V[pivotPos+1] < pivot){
		V[pivotPos] = V[pivotPos+1];
		V[pivotPos+1] = pivot;
		pivotPos++;
	}
	else{
		int tmp = V[rPos];
		V[rPos] = V[pivotPos+1];
		V[pivotPos+1] = tmp;
		rPos--;
	}

	myStackPush(l, r, pivotPos, rPos, pivot);
}

__global__ void renderY(uint8_t *yuv, uint32_t *bufferV, int H, int stackF, Stack *bufferS){
	int x = blockIdx.x, y = threadIdx.x;
	int index = x * blockDim.x + y;
	uint8_t tmp;
	if (H-1-x > bufferV[y]) tmp = 0;
	else tmp = 255;
	for(int i=0; i<=stackF; i++)
		if (y/2 == bufferS[i].pivotPos/2 && H-1-x <= bufferS[i].pivot) tmp = 76;
	yuv[index] = tmp;
}

__global__ void renderU(uint8_t *yuv, uint32_t *bufferV, int H, int stackF, Stack *bufferS){
	int x = blockIdx.x, y = threadIdx.x;
	int index = x * blockDim.x + y;
	uint8_t tmp = 128;
	for(int i=0; i<=stackF; i++)
		if (y == bufferS[i].pivotPos/2 && (H-1-x)*2 <= bufferS[i].pivot) tmp = 85;
	yuv[index] = tmp;
}

__global__ void renderV(uint8_t *yuv, uint32_t *bufferV, int H, int stackF, Stack *bufferS){
	int x = blockIdx.x, y = threadIdx.x;
	int index = x * blockDim.x + y;
	uint8_t tmp = 128;
	for(int i=0; i<=stackF; i++)
		if (y == bufferS[i].pivotPos/2 && (H-1-x)*2 <= bufferS[i].pivot) tmp = 255;
	yuv[index] = tmp;
}

void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	if (impl->t == 0)
		myPreprocessing();
	else
		for(int i=0; i<13; i++)
			myQuickSort();

	hipMemcpy(bufferV, V, W*sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(bufferStack, stack, (stackF+2)*sizeof(Stack), hipMemcpyHostToDevice);
	renderY<<<H, W>>>(yuv, bufferV, H, stackF, bufferStack);
	renderU<<<H/2, W/2>>>(yuv+W*H, bufferV, H/2, stackF, bufferStack);
	renderV<<<H/2, W/2>>>(yuv+W*H*5/4, bufferV, H/2, stackF, bufferStack);
	(impl->t)++;
}
