#include "lab1.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

struct Lab1VideoGenerator::Impl {
	int t = 0;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
	++(impl->t);
}
